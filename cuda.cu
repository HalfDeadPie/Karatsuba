#include "hip/hip_runtime.h"
#include "static_cuda.h"

// read polynomial from input file
TYPE* readPolynomial(const std::string &file, int *size){
    int degree;
    TYPE *result = nullptr;

    std::ifstream inputFile(file);
    if (inputFile) {
        inputFile >> degree;
        *size = degree;
        result = new TYPE[degree];
        for(int i=0; i<degree; i++){
            inputFile>>result[i];
        }
    }
    return result;
}

// print polynomial
void print(TYPE *A, int size){
    for (int i = 0; i < size ; i++) {
        PRINT<<A[i]<<" ";
    }
    PRINT<<std::endl;
}

/* computes the checksum based on equality of coefficients on random positions
 * but the result must be saved first. The best way how to save result is to
 * define SAVE in static.h and run the naive sequent algorithm*/
void checksum(std::string &file, const TYPE *polynomial, TYPE size){
    // THIS IS NOT REAL CHECKSUM

	int realSize;
    int mySize = 2 * size - 1;
    TYPE *realResult = readPolynomial(file, &realSize);

    // check size
	if(realSize != mySize)
        PRINT<<"\n\n!!!WRONG CHECKSUM (SIZE)!!!\n\n";
	
	// check first coefficients
	if(polynomial[0] != realResult[0])
		PRINT<<PRINT<<"\n\n!!!WRONG CHECKSUM (FIRST COEFFICIENT)!!!\n\n";
	
	// check last coefficients
	if(polynomial[mySize - 1] != realResult[mySize - 1]){
		PRINT<<"\n\n!!!WRONG CHECKSUM (LAST COEFFICIENT)!!!\n";
		PRINT<<polynomial[mySize - 1]<<" "<<realResult[mySize - 1]<<std::endl<<std::endl;
	}            
	
	// check even coefficient
	int position = 0;
  	position = mySize / 2;
  	if(position % 2 == 1) position++;
  	if(polynomial[position] != realResult[position] || polynomial[mySize-3] != realResult[mySize-3]){
  		 PRINT<<"\n\n!!!WRONG CHECKSUM (EVEN COEFFICIENT)!!!\n\n";
  	}
        
	// check random coefficients
    for(int i = 0; i < NUM_OF_CHECKS; i++){
        position = rand() % mySize;
        if(polynomial[position] != realResult[position])
            PRINT<<"\n\n!!!WRONG CHECKSUM (RESULT)!!!\n\n";
    }
}

// print the time resuls to standard output and times-file output
void printStats(double elapsed, double computingTime, const TYPE *polynomial, int size){
    std::string variable_parameter, name;

   	variable_parameter += "cuda_";
    
    #ifdef KARATSUBA
    name = "karatsuba_";
    #else
    name = "naive_";
    #endif

    #ifdef CHECKSUM
        std::string resultOutput = "results/result_" + std::to_string(size) + ".txt";
        checksum(resultOutput, polynomial, size);
    #endif

    if(!::outfile.is_open())
        ::outfile.open("times/" 
        + name 
        + variable_parameter 
        + std::to_string(size) 
        + "deg_"
        + std::to_string(::BLOCK_X) + "_times.txt");


    PRINT << "Total/computing time: " << elapsed << " / " << computingTime << std::endl;
    ::outfile<<::BLOCK_X<<"\t"<<computingTime<<"\t"<<elapsed<<::std::endl;
}


// computes the number of needed blocks depending on the threads and size
int blockRatio(int size, int threads){
    return ( (size + (threads - 1)) / threads );
}

//---------------------------------------------------------

__global__ void kernel_naive(TYPE *A, TYPE *B, TYPE *result, TYPE size_A, TYPE size_B, TYPE resultSize){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	
	if( i >= 0 && i < size_A) {
		if( j >= 0 && j < size_B ){
			atomicAdd( (result + i + j) , (A[i] * B[j]) ); 
		}
	}
}

// sequence version of naive algorithm
TYPE* cuda_naive(const TYPE *A, const TYPE *B, int size_A, int size_B, double *computingTime){
    int resultSize = size_A + size_B - 1;
    TYPE *__restrict__ hostResult = new TYPE[size_A + size_B - 1];
    
	int blockX = ::BLOCK_X;
	int blockY = ::BLOCK_Y;

	// blocks and grid initialisation
    dim3 block( blockX, blockY );
    dim3 grid( blockRatio(resultSize, blockX) , blockRatio(resultSize, blockY) );

    TYPE *__restrict__ devA, *__restrict__ devB, *__restrict__ devResult;
    hipMalloc((void**)&devA, size_A * sizeof(TYPE));
    hipMalloc((void**)&devB, size_B * sizeof(TYPE));
    hipMalloc((void**)&devResult, resultSize * sizeof(TYPE)); 
    
   	// hostA -> devA
	hipMemcpy(devA, A, size_A * sizeof(TYPE), 
		hipMemcpyHostToDevice);
	
	// hostB -> devB
	hipMemcpy(devB, B, size_B * sizeof(TYPE), 
		hipMemcpyHostToDevice);

	// start the timer without copying data
	auto start = std::chrono::high_resolution_clock::now();
	
	// K E R N E L S
	kernel_naive <<< grid, block >>> (devA, devB, devResult, size_A, size_B, resultSize);
	
	hipDeviceSynchronize();

	// Record end time without copying data
	auto finish = std::chrono::high_resolution_clock::now();
	        
	// Record end time
	std::chrono::duration<double> elapsed = finish - start;
	
	//PRINT<<"Elapsed time (no-copy): "<<elapsed.count()<<std::endl;
	*computingTime = elapsed.count();

	// devResult -> hostResult
	hipMemcpy(hostResult, devResult, resultSize * sizeof(TYPE), 
		hipMemcpyDeviceToHost);
	    
    return hostResult;
}

//---------------------------------------------------------

// compute the D according to iterative algorithm
__global__ void kernel_D(TYPE *A, TYPE *B, TYPE *D, TYPE size){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size)
    	D[i] = A[i] * B[i];
}

// set the first and last coefficients of the result
__global__ void kernel_res_bounds(TYPE *result, TYPE *D, TYPE size, TYPE resultSize){
	//result[resultSize - 1] = D[size - 1];
	result[0] = D[0];
	result[resultSize - 1] = D[size - 1];
}

// addition to even coefficients according to iterative algorithm
__global__ void kernel_res_even(TYPE *result, TYPE *D, TYPE resultSize){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i % 2 == 0 && i > 0 && i < resultSize - 1)
		atomicAdd(result + i, D[ i / 2 ]);
}

// the main computing of the result using for-loop (this seems to be faster)
__global__ void kernel_res_main(TYPE *A, TYPE *B, TYPE *D, TYPE *result, TYPE size, TYPE resultSize){
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if( i > 0 && i < resultSize - 1){
        TYPE start = (i >= size) ? (i % size ) + 1 : 0;
        TYPE end = (i + 1) / 2;
  
        for(TYPE inner = start; inner < end; inner++){
            atomicAdd(result + i, (  (A[inner] + A[i - inner]) * (B[inner] + B[i - inner]) ) );
            atomicSub(result + i, ( D[inner] + D[i-inner] ) );
        }		
	}
}


// the main computing of the result using grid (this seems to be slower)
__global__ void kernel_res_nested(TYPE *A, TYPE *B, TYPE *D, TYPE *result, TYPE size, TYPE resultSize){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;    
    
	if(i>=resultSize-1 || j>size) return;

	TYPE start = (i >= size) ? (i % size ) + 1 : 0;
    TYPE end = (i + 1) >> 1;
    
	if( i<resultSize-1 && j>=start && j<end ){
		atomicAdd(result+i, ( (A[j] + A[i - j] ) * ( B[j] + B[i - j]) ) );
		atomicSub(result+i,  ( D[j] + D[i - j] ) );
	}
	
}

//  iterative version of Karatsuba algorithm
TYPE* cuda_karatsuba(const TYPE *A, const TYPE *B, TYPE size, double *computingTime){
 	// compute the size of the result
	TYPE resultSize = 2 * size - 1;
	
	// the number of threads for one dimensional computing
	int threadsOne = ::NUM_THREADS;

	// dimension of blocks for grid
	int blockX = ::BLOCK_X;
	int blockY = ::BLOCK_Y;

	// blocks and grid initialisation
    dim3 block( blockX, blockY );
    dim3 grid( blockRatio(resultSize, blockX) , blockRatio(resultSize, blockY) );

	    
    // create empty coefficient vector with proper size and fill it with 0
    TYPE *__restrict__ hostResult = new TYPE[resultSize];
	TYPE *__restrict__ hostD = new TYPE[size];
	
	// create and allocate space for polynomials and the result
	TYPE *__restrict__ devA, *__restrict__ devB, *__restrict__ devD, *__restrict__ devResult;
	hipMalloc((void**)&devA, size * sizeof(TYPE));
	hipMalloc((void**)&devB, size * sizeof(TYPE));
	hipMalloc((void**)&devD, size * sizeof(TYPE));
	hipMalloc((void**)&devResult, resultSize * sizeof(TYPE));
	
	// hostA -> devA
	hipMemcpy(devA, A, size * sizeof(TYPE), 
		hipMemcpyHostToDevice);
	// hostB -> devB
	hipMemcpy(devB, B, size * sizeof(TYPE),
		hipMemcpyHostToDevice);

	// start the timer without copying data
	auto start = std::chrono::high_resolution_clock::now();

	// K E R N E L S	
	
	// compute the D
	kernel_D <<<blockRatio(size, threadsOne) , threadsOne>>> (devA, devB, devD, size);
	
	// ini the first and last coefficent of the result
	kernel_res_bounds <<<1, 1>>> (devResult, devD, size, resultSize);
	
	// add to even coefficients of result
	kernel_res_even <<<blockRatio(resultSize, threadsOne), threadsOne>>> (devResult, devD, resultSize);
	
	// compute the main part of the result 
	kernel_res_main <<<blockRatio(resultSize, threadsOne), threadsOne>>> (devA, devB, devD, devResult, size, resultSize);
	//kernel_res_nested <<<grid, block>>> (devA, devB, devD, devResult, size, resultSize);

	// wait for the kernels
	hipDeviceSynchronize();

	// Record end time without copying data
	auto finish = std::chrono::high_resolution_clock::now();
	        
	// Record end time
	std::chrono::duration<double> elapsed = finish - start;
	
	//PRINT<<"Elapsed time (no-copy): "<<elapsed.count()<<std::endl;
	*computingTime = elapsed.count();

	// devResult -> hostResult
	hipMemcpy(hostResult, devResult, resultSize * sizeof(TYPE), 
		hipMemcpyDeviceToHost);
	    
    return hostResult;
}


//---------------------------------------------------------


int main(int argc, char* argv[]) {

 	int size_A, size_B;
    TYPE *A = readPolynomial(argv[1], &size_A);
    TYPE *B = readPolynomial(argv[2], &size_B);
	double computingTime;

    // Record start time
    auto start = std::chrono::high_resolution_clock::now();

    // multiply the polynomials
    TYPE *result;

	// M U L T I P LI C A T I O N
	#ifdef KARATSUBA
    PRINT<<"Karatsuba version: CUDA"<<std::endl;
    result = cuda_karatsuba(A, B, size_A, &computingTime);
    #else
    PRINT<<"Naive version: CUDA"<<std::endl;
    result = cuda_naive(A, B, size_A, size_B, &computingTime);
    #endif

    // Record end time
    auto finish = std::chrono::high_resolution_clock::now();

    // Record end time
    std::chrono::duration<double> elapsed = finish - start;

    // print the result if DEBUG is defined
    #ifdef DEBUG
    print(result, 2 * size_A - 1);
    #endif

    printStats(elapsed.count(), computingTime, result, size_A);

    ::outfile.close();
    return 0;
}
